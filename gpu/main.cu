#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <sys/time.h>
#include "gputimer.h"
#define BLOCK_SIZE (1024)
double get_walltime()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return (double) (tp.tv_sec + tp.tv_usec*1e-6);
}
void Write(float* buffer, int np, char* output){
   FILE *f;
   f=fopen(output,"w");
    for(int i=0;i<np;++i){
       fprintf(f,"%15f \n",buffer[i]);
    }
    fclose(f);
}
void force_repulsion(int np, const float *pos, float L, float krepulsion, 
    float *forces)
{
    int i, j;
    float posi[4];
    float rvec[4];
    float s2, s, f;

    // initialize forces to zero
    for (i=0; i<3*np; i++)
        forces[i] = 0.;

    // loop over all pairs
    for (i=0; i<np; i++)
    {
        posi[0] = pos[3*i  ];
        posi[1] = pos[3*i+1];
        posi[2] = pos[3*i+2];

        for (j=i+1; j<np; j++)
        {
            // compute minimum image difference
            rvec[0] = remainder(posi[0] - pos[3*j  ], L);
            rvec[1] = remainder(posi[1] - pos[3*j+1], L);
            rvec[2] = remainder(posi[2] - pos[3*j+2], L);

            s2 = rvec[0]*rvec[0] + rvec[1]*rvec[1] + rvec[2]*rvec[2];

            if (s2 < 4)
            {
                s = sqrt(s2);
                rvec[0] /= s;
                rvec[1] /= s;
                rvec[2] /= s;
                f = krepulsion*(2.-s);

                forces[3*i  ] +=  f*rvec[0];
                forces[3*i+1] +=  f*rvec[1];
                forces[3*i+2] +=  f*rvec[2];
                forces[3*j  ] += -f*rvec[0];
                forces[3*j+1] += -f*rvec[1];
                forces[3*j+2] += -f*rvec[2];
            }
        }
    }
}

__global__ void gpu_find_repulsion(int np, float*pos, float L, float krepulsion, float* forces){
     int i = blockDim.x * blockIdx.x + threadIdx.x;
     if(i<np){
     	int j;
     	float posi[3];
     	float rvec[3];
     	float s2, s, f;
     	posi[0] = pos[3*i  ];
     	posi[1] = pos[3*i+1];
     	posi[2] = pos[3*i+2];
	 	for (j=i+1; j<np; ++j){
        // compute minimum image difference
         	rvec[0] = remainder(posi[0] - pos[3*j  ], L);
         	rvec[1] = remainder(posi[1] - pos[3*j+1], L);
         	rvec[2] = remainder(posi[2] - pos[3*j+2], L);
         	s2 = rvec[0]*rvec[0] + rvec[1]*rvec[1] + rvec[2]*rvec[2];
         	if (s2 < 4){
             	s = sqrt(s2);
             	rvec[0] /= s;
             	rvec[1] /= s;
             	rvec[2] /= s;
             	f = krepulsion*(2.-s);
             	forces[3*i  ] +=  f*rvec[0];
             	forces[3*i+1] +=  f*rvec[1];
                forces[3*i+2] +=  f*rvec[2];
             	atomicAdd(&forces[3*j],-f*rvec[0]);
             	atomicAdd(&forces[3*j+1],-f*rvec[1]);
             	atomicAdd(&forces[3*j+2],-f*rvec[2]);
            }
        }
     }
}


int main(int argc, char *argv[])
{
    int i;
    int np = 100;             // default number of particles
    float phi = 0.3;         // volume fraction
    float krepulsion = 125.; // force constant
    float *pos;
    float *forces;
    double time0, time1;

    if (argc > 1)
        np = atoi(argv[1]);

    // compute simulation box width
    float L = pow(4./3.*3.1415926536*np/phi, 1./3.);

    // generate random particle positions inside simulation box
    forces = (float *) malloc(3*np*sizeof(float));
    pos    = (float *) malloc(3*np*sizeof(float));
    for (i=0; i<3*np; i++)
        pos[i] = rand()/(float)RAND_MAX*L;
    time0 = get_walltime();
    force_repulsion(np, pos, L, krepulsion, forces);
    time1 = get_walltime();
    //print performance and write to file
    printf("number of particles: %d\n", np);
    printf("elapsed time of cpu program: %f seconds\n", time1-time0);
    Write(forces,3*np,"cpu_output"); 
    //reinitialization of forces
    // for(int i=0;i<np*3;++i) forces[i]=0.;
    //gpu program
    float *gpu_pos;
    float *gpu_forces;
    int bytes=3*np*sizeof(float);
    GpuTimer timer;
    hipMalloc((void**)&gpu_pos,bytes);
    hipMalloc((void**)&gpu_forces,bytes);
    hipMemcpy(gpu_pos, pos, bytes, hipMemcpyHostToDevice);
    hipMemset(gpu_forces, 0, bytes);
    timer.Start();
    gpu_find_repulsion<<<(3*np+BLOCK_SIZE-1)/BLOCK_SIZE,BLOCK_SIZE>>>(np, gpu_pos, L, krepulsion, gpu_forces);
    timer.Stop();
    hipMemcpy(forces, gpu_forces, bytes, hipMemcpyDeviceToHost);
    printf("number of particles: %d\n", np);
    printf("elapsed time of gpu program: %f seconds\n", timer.Elapsed()/1000);
    Write(forces,3*np,"gpu_output");
    printf("speed up of gpu is %f \n",(time1-time0)/(timer.Elapsed()/1000));
    hipFree(gpu_pos);
    hipFree(gpu_forces);
    free(forces);
    free(pos);

    return 0;
}
